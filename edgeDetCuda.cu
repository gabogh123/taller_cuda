#include <iostream>
#include <fstream>
#include <vector>
#include <chrono>
#include <hip/hip_runtime.h>

using namespace std;
using namespace std::chrono;

// Kernel CUDA para aplicar el filtro de detección de bordes
__global__ void edgeDetectionCUDA(const unsigned char *src, unsigned char *dst, int width, int height) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    // Definir los kernels de Sobel para X e Y
    int gx[3][3] = {{-1, 0, 1}, {-2, 0, 2}, {-1, 0, 1}};
    int gy[3][3] = {{-1, -2, -1}, {0, 0, 0}, {1, 2, 1}};

    // Asegurarse de que el hilo esté dentro de los límites de la imagen
    if (x > 0 && x < width - 1 && y > 0 && y < height - 1) {
        int sumX = 0, sumY = 0;
        // Aplicar los kernels de Sobel
        for (int i = -1; i <= 1; i++) {
            for (int j = -1; j <= 1; j++) {
                int pixel = src[(y + i) * width + (x + j)];
                sumX += pixel * gx[i + 1][j + 1];
                sumY += pixel * gy[i + 1][j + 1];
            }
        }
        // Calcular la magnitud del gradiente
        int sum = abs(sumX) + abs(sumY);
        // Asignar el valor al píxel de salida (clamp a 255)
        dst[y * width + x] = (sum > 255) ? 255 : sum;
    }
}

// Función envolvente para llamar al kernel CUDA
void edgeDetectionCUDAWrapper(unsigned char *src, unsigned char *dst, int width, int height) {
    unsigned char *d_src, *d_dst;
    size_t size = width * height * sizeof(unsigned char);

    // Reservar memoria en el dispositivo (GPU)
    hipError_t err;
    err = hipMalloc(&d_src, size);
    if (err != hipSuccess) {
        cerr << "Error al reservar memoria en el dispositivo: " << hipGetErrorString(err) << endl;
        return;
    }

    err = hipMalloc(&d_dst, size);
    if (err != hipSuccess) {
        cerr << "Error al reservar memoria en el dispositivo: " << hipGetErrorString(err) << endl;
        hipFree(d_src);
        return;
    }

    // Copiar datos de la imagen desde el host (CPU) al dispositivo (GPU)
    err = hipMemcpy(d_src, src, size, hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        cerr << "Error al copiar datos al dispositivo: " << hipGetErrorString(err) << endl;
        hipFree(d_src);
        hipFree(d_dst);
        return;
    }

    // Definir la configuración de los bloques e hilos
    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((width + threadsPerBlock.x - 1) / threadsPerBlock.x, (height + threadsPerBlock.y - 1) / threadsPerBlock.y);

    // Llamar al kernel CUDA
    edgeDetectionCUDA<<<numBlocks, threadsPerBlock>>>(d_src, d_dst, width, height);
    err = hipGetLastError();
    if (err != hipSuccess) {
        cerr << "Error en el kernel: " << hipGetErrorString(err) << endl;
    }

    // Copiar los resultados desde el dispositivo (GPU) al host (CPU)
    err = hipMemcpy(dst, d_dst, size, hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        cerr << "Error al copiar datos del dispositivo: " << hipGetErrorString(err) << endl;
    }

    // Liberar la memoria en el dispositivo (GPU)
    hipFree(d_src);
    hipFree(d_dst);
}

// Función para leer la imagen desde un archivo de texto
bool readImageFromText(const string& filename, vector<unsigned char>& image, int& width, int& height) {
    ifstream file(filename);
    if (!file.is_open()) {
        cout << "Error al abrir el archivo de texto" << endl;
        return false;
    }

    file >> height >> width;
    image.resize(width * height);

    for (int y = 0; y < height; y++) {
        for (int x = 0; x < width; x++) {
            int pixel;
            file >> pixel;
            image[y * width + x] = static_cast<unsigned char>(pixel);
        }
    }

    file.close();
    return true;
}

// Función para guardar la imagen en un archivo de texto
void saveImageToText(const string& filename, const vector<unsigned char>& image, int width, int height) {
    ofstream file(filename);
    file << height << " " << width << endl;
    for (int y = 0; y < height; y++) {
        for (int x = 0; x < width; x++) {
            file << static_cast<int>(image[y * width + x]) << " ";
        }
        file << endl;
    }
    file.close();
}

int main() {
    // Leer la imagen desde un archivo de texto
    vector<unsigned char> image;
    int width, height;
    if (!readImageFromText("imagen5.txt", image, width, height)) {
        return -1;
    }

    // Imagen de salida
    vector<unsigned char> edgeImage(width * height);

    // Medir el tiempo de ejecución
    auto start = high_resolution_clock::now();
    edgeDetectionCUDAWrapper(image.data(), edgeImage.data(), width, height);
    auto stop = high_resolution_clock::now();
    auto duration = duration_cast<milliseconds>(stop - start);

    // Mostrar el tiempo de ejecución
    cout << "Tiempo de ejecución: " << duration.count() << " ms" << endl;

    // Guardar la imagen resultante en un archivo de texto
    saveImageToText("imagen_bordes_cuda.txt", edgeImage, width, height);

    return 0;
}

